
#include "hip/hip_runtime.h"


#include <stdio.h>

// It's a M * N matrix
#define M 6
#define N 3

// Each element is computed on one thread

__global__ void add (int *A, int *B, int *C) {
    // Get the 1D Array index of the matrix
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    printf("idx = %d\n", idx);
    C[idx] = A[idx] + B[idx];
}

int main () {
    
    // Host copies of the variables
    int A[M * N], B[M * N], C[M * N];

    int i, j;
    for (i = 0; i < M * N; ++i) {
        A[i] = i + 1;
        B[i] = M * N - i - 1;
    }

    // Device copies of the variables
    int *d_a, *d_b, *d_c;

    int size = sizeof(int) * M * N;

    // Allocate memories to device copies of the objects
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    // Copy inputs to device
    hipMemcpy(d_a, &A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &B, size, hipMemcpyHostToDevice);

    // Launch kernel onto the device
    add<<<M, N>>>(d_a, d_b, d_c);

    // Copy the result back to the host
    hipMemcpy(&C, d_c, size, hipMemcpyDeviceToHost);

    // Outpoooot it
    printf("A:\n");
    for (i = 0; i < N; ++i) {
        for (j = 0; j < M; ++j) {
            printf("%d\t", A[i * M + j]);
        }
        printf("\n");
    }
    printf("\n");

    printf("B:\n");
    for (i = 0; i < N; ++i) {
        for (j = 0; j < M; ++j) {
            printf("%d\t", B[i * M + j]);
        }
        printf("\n");
    }
    printf("\n");

    printf("A + B:\n");
    for (i = 0; i < N; ++i) {
        for (j = 0; j < M; ++j) {
            printf("%d\t", C[i * M + j]);
        }
        printf("\n");
    }
    printf("\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

	getchar();

    return 0;
}